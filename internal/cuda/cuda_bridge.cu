#include "hip/hip_runtime.h"
// Codi  per a la interacció Go <-> C.
// CreatedAt: 2024/11/24 dg. JIQ

#include "cuda_bridge.h"
#include "fixed_point.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Declaració del kernel
extern __global__ void process_kernel(fixed_point_t* input, fixed_point_t* output, int size);

extern "C" {

int cuda_init(void) {
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }
    return 0;
}

void cuda_cleanup(void) {
    hipDeviceReset();
}

int cuda_process_fixed_point(int32_t* input, int32_t* output, int size) {
    fixed_point_t *d_input, *d_output;
    hipError_t err;
    
    // Aloca memòria a la GPU
    err = hipMalloc(&d_input, size * sizeof(fixed_point_t));
    if (err != hipSuccess) {
        printf("CUDA malloc input error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    err = hipMalloc(&d_output, size * sizeof(fixed_point_t));
    if (err != hipSuccess) {
        hipFree(d_input);
        printf("CUDA malloc output error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Copia input a la GPU
    err = hipMemcpy(d_input, input, size * sizeof(fixed_point_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_input);
        hipFree(d_output);
        printf("CUDA memcpy input error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Executa el kernel
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    process_kernel<<<numBlocks, blockSize>>>(d_input, d_output, size);
    
    // Comprova errors del kernel
    err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_input);
        hipFree(d_output);
        printf("CUDA kernel error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Copia resultat de tornada
    err = hipMemcpy(output, d_output, size * sizeof(fixed_point_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        hipFree(d_input);
        hipFree(d_output);
        printf("CUDA memcpy output error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Neteja
    hipFree(d_input);
    hipFree(d_output);
    
    return 0;
}

}